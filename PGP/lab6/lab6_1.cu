#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

using namespace std;

#define CUDA_CHECK_RETURN(value)                                          \
    {                                                                     \
        hipError_t _m_cudaStat = value;                                  \
        if (_m_cudaStat != hipSuccess)                                   \
        {                                                                 \
            fprintf(stderr, "Error %s at line %d in file %s\n",           \
                    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)
__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main(void)
{
    hipDeviceProp_t prop;
    int whichDevice;
    CUDA_CHECK_RETURN(hipGetDevice(&whichDevice));
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, whichDevice));
    if (!prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no "
               "speed up from streams\n");
        return 0;
    }

    hipEvent_t start, stop;
    float elapsedTime;
    // start the timers
    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));
    CUDA_CHECK_RETURN(hipEventRecord(start, 0));

    // initialize the stream
    hipStream_t stream;
    CUDA_CHECK_RETURN(hipStreamCreate(&stream));

    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_a,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_b,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c,
                                 N * sizeof(int)));

    // allocate page-locked memory, used to stream
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_a,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_b,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_c,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N)
    {
        // copy the locked memory to the device, async
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_a, host_a + i,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream));
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_b, host_b + i,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream));
        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
        // copy the data from device to locked memory
        CUDA_CHECK_RETURN(hipMemcpyAsync(host_c + i, dev_c,
                                          N * sizeof(int),
                                          hipMemcpyDeviceToHost,
                                          stream));
    }

    // copy result chunk from locked to full buffer
    CUDA_CHECK_RETURN(hipStreamSynchronize(stream));

    CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime,
                                           start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    // cleanup the streams and memory
    CUDA_CHECK_RETURN(hipHostFree(host_a));
    CUDA_CHECK_RETURN(hipHostFree(host_b));
    CUDA_CHECK_RETURN(hipHostFree(host_c));
    CUDA_CHECK_RETURN(hipFree(dev_a));
    CUDA_CHECK_RETURN(hipFree(dev_b));
    CUDA_CHECK_RETURN(hipFree(dev_c));

    CUDA_CHECK_RETURN(hipStreamDestroy(stream));
    return 0;
}