#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

using namespace std;

#define CUDA_CHECK_RETURN(value)                                          \
    {                                                                     \
        hipError_t _m_cudaStat = value;                                  \
        if (_m_cudaStat != hipSuccess)                                   \
        {                                                                 \
            fprintf(stderr, "Error %s at line %d in file %s\n",           \
                    hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
            exit(1);                                                      \
        }                                                                 \
    }

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)
__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}

int main()
{
    hipDeviceProp_t prop;
    int whichDevice;
    CUDA_CHECK_RETURN(hipGetDevice(&whichDevice));
    CUDA_CHECK_RETURN(hipGetDeviceProperties(&prop, whichDevice));

    if (!prop.deviceOverlap)
    {
        printf("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }
    hipEvent_t start, stop;
    float elapsedTime;
    // start the timers
    CUDA_CHECK_RETURN(hipEventCreate(&start));
    CUDA_CHECK_RETURN(hipEventCreate(&stop));
    CUDA_CHECK_RETURN(hipEventRecord(start, 0));

    // initialize the streams
    hipStream_t stream0, stream1;
    CUDA_CHECK_RETURN(hipStreamCreate(&stream0));
    CUDA_CHECK_RETURN(hipStreamCreate(&stream1));

    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0; // GPU buffers for stream0
    int *dev_a1, *dev_b1, *dev_c1; // GPU buffers for stream1
    // allocate the memory on the GPU
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_a0,
                                 N * sizeof(int)));

    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_b0,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c0,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_a1,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_b1,
                                 N * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&dev_c1,
                                 N * sizeof(int)));
    // allocate page-locked memory, used to stream
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_a,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_b,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    CUDA_CHECK_RETURN(hipHostAlloc((void **)&host_c,
                                    FULL_DATA_SIZE * sizeof(int),
                                    hipHostMallocDefault));
    for (int i = 0; i < FULL_DATA_SIZE; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // now loop over full data, in bite-sized chunks
    for (int i = 0; i < FULL_DATA_SIZE; i += N * 2)
    {
        // enqueue copies of a in stream0 and stream1
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_a0, host_a + i,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream0));
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_a1, host_a + i + N,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream1));
        // enqueue copies of b in stream0 and stream1
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_b0, host_b + i,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream0));
        CUDA_CHECK_RETURN(hipMemcpyAsync(dev_b1, host_b + i + N,
                                          N * sizeof(int),
                                          hipMemcpyHostToDevice,
                                          stream1));
        // enqueue kernels in stream0 and stream1
        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
        // enqueue copies of c from device to locked memory
        CUDA_CHECK_RETURN(hipMemcpyAsync(host_c + i, dev_c0,
                                          N * sizeof(int),
                                          hipMemcpyDeviceToHost,
                                          stream0));
        CUDA_CHECK_RETURN(hipMemcpyAsync(host_c + i + N, dev_c1,
                                          N * sizeof(int),
                                          hipMemcpyDeviceToHost,
                                          stream1));
    }

    CUDA_CHECK_RETURN(hipStreamSynchronize(stream0));
    CUDA_CHECK_RETURN(hipStreamSynchronize(stream1));

    CUDA_CHECK_RETURN(hipEventRecord(stop, 0));
    CUDA_CHECK_RETURN(hipEventSynchronize(stop));
    CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime,
                                           start, stop));
    printf("Time taken: %3.1f ms\n", elapsedTime);
    // cleanup the streams and memory
    CUDA_CHECK_RETURN(hipHostFree(host_a));
    CUDA_CHECK_RETURN(hipHostFree(host_b));
    CUDA_CHECK_RETURN(hipHostFree(host_c));

    CUDA_CHECK_RETURN(hipFree(dev_a0));
    CUDA_CHECK_RETURN(hipFree(dev_b0));
    CUDA_CHECK_RETURN(hipFree(dev_c0));
    CUDA_CHECK_RETURN(hipFree(dev_a1));
    CUDA_CHECK_RETURN(hipFree(dev_b1));
    CUDA_CHECK_RETURN(hipFree(dev_c1));
    CUDA_CHECK_RETURN(hipStreamDestroy(stream0));
    CUDA_CHECK_RETURN(hipStreamDestroy(stream1));

    return 0;
}