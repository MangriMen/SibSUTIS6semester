
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void saxpy(int n, float a, float *__restrict x, float *__restrict y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        y[i] = a * x[i] + y[i];
    }
}

int main()
{
    const int n = 1 << 20;

    hipEvent_t startGlobal, stopGlobal, startLocal, stopLocal;
    hipEventCreate(&startGlobal);
    hipEventCreate(&stopGlobal);
    hipEventCreate(&startLocal);
    hipEventCreate(&stopLocal);

    hipEventRecord(startGlobal);

    float *hostArr = (float *)malloc(n * sizeof(float));
    float *hostArrRes = (float *)malloc(n * sizeof(float));

    float *cudaArr = NULL;
    float *cudaArrRes = NULL;
    hipMalloc((void **)&cudaArr, (n) * sizeof(float));
    hipMalloc((void **)&cudaArrRes, (n) * sizeof(float));

    for (int i = 0; i < n; i++)
    {
        hostArr[i] = i;
    }

    hipMemcpy(cudaArr, hostArr, n * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(startLocal);
    saxpy<<<4096, 256>>>(n, 2.0, cudaArr, cudaArrRes);
    hipEventRecord(stopLocal);
    hipEventSynchronize(stopLocal);

    hipMemcpy(hostArrRes, cudaArrRes + 1, n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stopGlobal);
    hipEventSynchronize(stopGlobal);

    float globalElapsedTime = 0;
    hipEventElapsedTime(&globalElapsedTime, startGlobal, stopGlobal);

    float localElapsedTime = 0;
    hipEventElapsedTime(&localElapsedTime, startLocal, stopLocal);

    printf("Native\n\tGlobal: %f ms\n\tLocal: %f ms\n", globalElapsedTime, localElapsedTime);

    hipEventDestroy(startGlobal);
    hipEventDestroy(stopGlobal);
    hipEventDestroy(startLocal);
    hipEventDestroy(stopLocal);

    free(hostArr);
    free(hostArrRes);
    free(cudaArr);
    free(cudaArrRes);

    return EXIT_SUCCESS;
}