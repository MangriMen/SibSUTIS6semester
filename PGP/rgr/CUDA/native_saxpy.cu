
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void saxpy(int n, float a, float *__restrict x, float *__restrict y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        y[i] = a * x[i] + y[i];
    }
}

int main()
{
    const int n = 1 << 24;

    hipEvent_t startGlobal, endGlobal, startLocal, endLocal;
    hipEventCreate(&startGlobal);
    hipEventCreate(&endGlobal);
    hipEventCreate(&startLocal);
    hipEventCreate(&endLocal);

    hipEventRecord(startGlobal);

    float *hostArr = (float *)malloc(n * sizeof(float));
    float *hostArrRes = (float *)malloc(n * sizeof(float));

    float *cudaArr = NULL;
    hipMalloc((void **)&cudaArr, (n) * sizeof(float));

    float *cudaArrRes = NULL;
    hipMalloc((void **)&cudaArrRes, (n) * sizeof(float));

    for (int i = 0; i < n; i++)
    {
        hostArr[i] = i;
    }

    hipMemcpy(cudaArr, hostArr, n * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(startLocal);
    saxpy<<<4096, 256>>>(n, 2.0, cudaArr, cudaArrRes);
    hipEventRecord(endLocal);
    hipEventSynchronize(endLocal);

    hipMemcpy(hostArrRes, cudaArrRes + 1, n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(endGlobal);
    hipEventSynchronize(endGlobal);

    float globalElapsedTime = 0;
    hipEventElapsedTime(&globalElapsedTime, startGlobal, endGlobal);

    float localElapsedTime = 0;
    hipEventElapsedTime(&localElapsedTime, startLocal, endLocal);

    printf("Native\n\tGlobal: %f ms\n\tLocal: %f ms\n", globalElapsedTime, localElapsedTime);

    hipEventDestroy(startGlobal);
    hipEventDestroy(endGlobal);
    hipEventDestroy(startLocal);
    hipEventDestroy(endLocal);

    free(hostArr);
    free(hostArrRes);
    free(cudaArr);
    free(cudaArrRes);

    return EXIT_SUCCESS;
}