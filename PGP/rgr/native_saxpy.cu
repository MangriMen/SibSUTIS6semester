
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <thread>
#include <chrono>

using namespace std;

__global__ void saxpy(int n, float a, float *__restrict x, float *__restrict y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        y[i] = a * x[i] + y[i];
    }
}

int main(int argc, char *argv[])
{
    int n = 1 << 24;

    if (argc > 1)
    {
        n = stoi(argv[1]);
    }
    else
    {
        return EXIT_FAILURE;
    }

    hipEvent_t startGlobal, endGlobal, startLocal, endLocal;
    hipEventCreate(&startGlobal);
    hipEventCreate(&endGlobal);
    hipEventCreate(&startLocal);
    hipEventCreate(&endLocal);

    hipEventRecord(startGlobal);

    float *hostArr = (float *)malloc(n * sizeof(float));
    float *hostArrRes = (float *)malloc(n * sizeof(float));

    float *cudaArr = NULL;
    hipMalloc((void **)&cudaArr, (n + 1) * sizeof(float));

    float *cudaArrRes = NULL;
    hipMalloc((void **)&cudaArrRes, (n + 1) * sizeof(float));

    for (int i = 0; i < n; i++)
    {
        hostArr[i] = i;
    }

    hipMemcpy(cudaArr, hostArr, n * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(startLocal);
    saxpy<<<512, 32>>>(n, 2.0, cudaArr, cudaArrRes);
    hipEventRecord(endLocal);
    hipEventSynchronize(endLocal);

    hipMemcpy(hostArrRes, cudaArrRes + 1, n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(endGlobal);
    hipEventSynchronize(endGlobal);

    float globalElapsedTime = 0;
    hipEventElapsedTime(&globalElapsedTime, startGlobal, endGlobal);

    float localElapsedTime = 0;
    hipEventElapsedTime(&localElapsedTime, startLocal, endLocal);

    printf("Native\n\tGlobal: %f ms\n\tLocal: %f ms\n", globalElapsedTime, localElapsedTime);

    ofstream fileOut("result.csv", ios::app);
    fileOut << "native;" << globalElapsedTime << ";" << localElapsedTime << "\n";
    fileOut.close();

    hipEventDestroy(startGlobal);
    hipEventDestroy(endGlobal);
    hipEventDestroy(startLocal);
    hipEventDestroy(endLocal);

    hipFree(cudaArr);
    hipFree(cudaArrRes);

    free(hostArr);
    free(hostArrRes);

    return EXIT_SUCCESS;
}