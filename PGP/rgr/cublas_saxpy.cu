#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <fstream>

using namespace std;

int main()
{
    hipEvent_t startGlobal;
    hipEvent_t endGlobal;
    hipEvent_t startLocal;
    hipEvent_t endLocal;

    hipEventCreate(&startGlobal);
    hipEventCreate(&endGlobal);
    hipEventCreate(&startLocal);
    hipEventCreate(&endLocal);

    hipEventRecord(startGlobal);

    const int n = 1 << 24;
    const size_t size_in_bytes = (n * sizeof(float));

    float *A_dev = NULL;
    hipMalloc((void **)&A_dev, size_in_bytes);

    float *B_dev = NULL;
    hipMalloc((void **)&B_dev, size_in_bytes);

    float *A_h = NULL;
    hipHostMalloc((void **)&A_h, size_in_bytes);

    float *B_h = NULL;
    hipHostMalloc((void **)&B_h, size_in_bytes);

    memset(A_h, 0, size_in_bytes);
    memset(B_h, 0, size_in_bytes);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    for (int i = 0; i < n; i++)
    {
        A_h[i] = (float)i;
    }

    const int num_rows = n;
    const int num_cols = 1;
    const size_t elem_size = sizeof(float);

    //Копирование матрицы с числом строк num_elem и одним столбцом с
    //хоста на устройство
    hipblasSetMatrix(num_rows, num_cols, elem_size, A_h,
                    num_rows, A_dev, num_rows);

    //Очищаем массив на устройстве
    hipMemset(B_dev, 0, size_in_bytes);

    // выполнение SingleAlphaXPlusY
    const int stride = 1;
    float alpha = 2.0F;
    hipEventRecord(startLocal);

    hipblasSaxpy(cublas_handle, n, &alpha, A_dev,
                stride, B_dev, stride);

    hipEventRecord(endLocal);

    //Копирование матриц с числом строк num_elem и одним столбцом с
    //устройства на хост
    hipblasGetMatrix(num_rows, num_cols, elem_size, A_dev,
                    num_rows, A_h, num_rows);
    hipblasGetMatrix(num_rows, num_cols, elem_size, B_dev,
                    num_rows, B_h, num_rows);

    // Удостоверяемся, что все асинхронные вызовы выполнены
    const int default_stream = 0;
    hipStreamSynchronize(default_stream);

    hipblasDestroy(cublas_handle);
    hipFree(A_dev);
    hipFree(B_dev);

    hipHostFree(A_h);
    hipHostFree(A_h);
    hipHostFree(B_h);

    hipEventRecord(endGlobal);

    float globalElapsedTime = 0;
    hipEventElapsedTime(&globalElapsedTime, startGlobal, endGlobal);

    float localElapsedTime = 0;
    hipEventElapsedTime(&localElapsedTime, startLocal, endLocal);

    printf("CUBLAS\n\tGlobal: %f ms\n\tLocal: %f ms\n", globalElapsedTime, localElapsedTime);

    ofstream fileOut("result.csv", ios::app);
    fileOut << "cublas;" << globalElapsedTime << ";" << localElapsedTime << "\n";
    fileOut.close();

    hipEventDestroy(startGlobal);
    hipEventDestroy(endGlobal);
    hipEventDestroy(startLocal);
    hipEventDestroy(endLocal);

    hipDeviceReset();

    return EXIT_SUCCESS;
}