
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <hipfft/hipfft.h>

using namespace std;

#define NX 365
#define BATCH 1

int main()
{
    hipfftHandle plan;
    hipfftComplex *data;

    hipfftComplex *data_h = new hipfftComplex[NX * BATCH];

    ifstream fileIn("w1991.dat");

    if (!fileIn.is_open())
    {
        cerr << "Failed to open in file." << endl;
        return EXIT_FAILURE;
    }

    float month = 0;
    float day = 0;
    float wolf = 0;
    float temp = 0;
    for (int i = 0; i < NX; i++)
    {
        fileIn >> month >> day >> wolf >> temp;
        if (wolf != 999)
        {
            data_h[i].x = wolf;
        }
        else
        {
            data_h[i].x = 0.0f;
        }

        data_h[i].y = 0.0f;
    }

    fileIn.close();

    hipMalloc((void **)&data, sizeof(hipfftComplex) * NX * BATCH);
    hipMemcpy(data, data_h, sizeof(hipfftComplex) * NX * BATCH, hipMemcpyHostToDevice);

    if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS)
    {
        cerr << "CUFFT error: Plan creation failed." << endl;
        return -1;
    }
    if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
        cerr << "CUFFT error: ExecC2C Forward failed." << endl;
        return -1;
    }

    hipDeviceSynchronize();

    hipMemcpy(data_h, data, NX * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    ofstream fileOut("out.dat");
    if (!fileOut.is_open())
    {
        cerr << "Failed to open out file." << endl;
        return EXIT_FAILURE;
    }

    for (int i = 0; i < NX; i++)
    {
        cout << setw(-20) << data_h[i].x << setw(20) << data_h[i].y << endl;
        fileOut << data_h[i].x << ";" << data_h[i].y << endl;
    }

    fileOut.close();

    hipfftDestroy(plan);
    hipFree(data);
    delete[] data_h;

    return EXIT_SUCCESS;
}