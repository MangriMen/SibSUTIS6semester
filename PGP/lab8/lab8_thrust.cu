#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

struct saxpy_functor
{
    const float a;
    saxpy_functor(float _a) : a(_a) {}
    __host__ __device__ float operator()(float x, float y)
    {
        return a * x + y;
    }
};

void saxpy(float a, thrust::device_vector<float> &x,
           thrust::device_vector<float> &y)
{
    saxpy_functor func(a);
    thrust::transform(x.begin(), x.end(), y.begin(), y.begin(), func);
}

int main()
{
    hipEvent_t startAll;
    hipEvent_t startFunc;
    hipEvent_t endFunc;
    hipEvent_t endAll;

    hipEventCreate(&startAll);
    hipEventCreate(&startFunc);
    hipEventCreate(&endFunc);
    hipEventCreate(&endAll);

    hipEventRecord(startAll);

    thrust::host_vector<float> h1(1 << 24);
    thrust::host_vector<float> h2(1 << 24);
    thrust::sequence(h1.begin(), h1.end());
    thrust::fill(h2.begin(), h2.end(), 0);
    thrust::device_vector<float> d1 = h1;
    thrust::device_vector<float> d2 = h2;

    hipEventRecord(startFunc);

    saxpy(2.0F, d1, d2);

    hipEventRecord(endFunc);

    h2 = d2;
    h1 = d1;

    hipEventRecord(endAll);

    for (int i = 0; i < (1 << 8); i++)
    {
        printf("%g\t%g\n", h1[i], h2[i]);
    }

    float func_time = 0;
    hipEventElapsedTime(&func_time, startFunc, endFunc);

    float program_time = 0;
    hipEventElapsedTime(&program_time, startAll, endAll);

    printf("SAXPY: %f\n", func_time);
    printf("Program: %f\n", program_time);

    hipEventDestroy(startFunc);
    hipEventDestroy(startAll);
    hipEventDestroy(endFunc);
    hipEventDestroy(endAll);

    return 0;
}