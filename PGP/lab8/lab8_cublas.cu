#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

int main()
{
    hipEvent_t startAll;
    hipEvent_t startFunc;
    hipEvent_t endFunc;
    hipEvent_t endAll;

    hipEventCreate(&startAll);
    hipEventCreate(&startFunc);
    hipEventCreate(&endFunc);
    hipEventCreate(&endAll);

    hipEventRecord(startAll);

    const int num_elem = 1 << 24;
    const size_t size_in_bytes = (num_elem * sizeof(float));

    float *A_dev;
    hipMalloc((void **)&A_dev, size_in_bytes);

    float *B_dev;
    hipMalloc((void **)&B_dev, size_in_bytes);

    float *A_h;
    hipHostMalloc((void **)&A_h, size_in_bytes);

    float *B_h;
    hipHostMalloc((void **)&B_h, size_in_bytes);

    memset(A_h, 0, size_in_bytes);
    memset(B_h, 0, size_in_bytes);

    // Инициализация библиотеки CUBLAS
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    for (int i = 0; i < num_elem; i++)
    {
        A_h[i] = (float)i;
    }

    const int num_rows = num_elem;
    const int num_cols = 1;
    const size_t elem_size = sizeof(float);

    //Копирование матрицы с числом строк num_elem и одним столбцом с
    //хоста на устройство
    hipblasSetMatrix(num_rows, num_cols, elem_size, A_h,
                    num_rows, A_dev, num_rows);

    //Очищаем массив на устройстве
    hipMemset(B_dev, 0, size_in_bytes);

    // выполнение SingleAlphaXPlusY
    const int stride = 1;
    float alpha = 2.0F;
    hipEventRecord(startFunc);

    hipblasSaxpy(cublas_handle, num_elem, &alpha, A_dev,
                stride, B_dev, stride);

    hipEventRecord(endFunc);

    //Копирование матриц с числом строк num_elem и одним столбцом с
    //устройства на хост
    hipblasGetMatrix(num_rows, num_cols, elem_size, A_dev,
                    num_rows, A_h, num_rows);
    hipblasGetMatrix(num_rows, num_cols, elem_size, B_dev,
                    num_rows, B_h, num_rows);

    // Удостоверяемся, что все асинхронные вызовы выполнены
    const int default_stream = 0;
    hipStreamSynchronize(default_stream);

    hipblasDestroy(cublas_handle);
    hipFree(A_dev);
    hipFree(B_dev);

    hipHostFree(A_h);
    hipHostFree(A_h);
    hipHostFree(B_h);

    hipEventRecord(endAll);

    float func_time = 0;
    hipEventElapsedTime(&func_time, startFunc, endFunc);

    float program_time = 0;
    hipEventElapsedTime(&program_time, startAll, endAll);

    printf("SAXPY: %f\n", func_time);
    printf("Program: %f\n", program_time);

    hipEventDestroy(startFunc);
    hipEventDestroy(startAll);
    hipEventDestroy(endFunc);
    hipEventDestroy(endAll);

    hipDeviceReset();

    return 0;
}